#include "hip/hip_runtime.h"
#include <iostream>
#include "Eigen/Dense"
#include <vector>
#include <cstdlib>
#include <fstream>
#include <chrono>
#include <hip/hip_runtime.h>

#define ROW1 5000
#define COL1 5000
#define COL2 5000
#define ROW2 5000

#define BLOCK_SIZE  32 


using namespace std;

// Get a matrix element
__device__ int GetElement(const int* elements, int stride, int row, int col) {
    return elements[row * stride + col];
}

// Set a matrix element
__device__ void SetElement(int* elements, int stride, int row, int col, int value) {
    elements[row * stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
__device__ void GetSubMatrix(const int* elements, int* subElements, int stride, int blockRow, int blockCol, int row, int col) {
    int globalRow = blockRow * BLOCK_SIZE + row;
    int globalCol = blockCol * BLOCK_SIZE + col;
    if (globalRow < stride && globalCol < stride) {
        subElements[row * BLOCK_SIZE + col] = elements[globalRow * stride + globalCol];
    } else {
        subElements[row * BLOCK_SIZE + col] = 0;
    }
}


__global__ void matrixMultiply(int *A, int *B, int *C, int numARows, int numACols, int numBCols, size_t pitchA, size_t pitchB, size_t pitchC)
{

 #ifdef USE_SHARED_MEMORY
    // Block row and column
        int blockRow = blockIdx.y;
        int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
        __shared__ int As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ int Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
        int Cvalue = 0;

    // Thread row and column within Csub
        int row = threadIdx.y;
        int col = threadIdx.x;

        for (int m = 0; m < (numACols + BLOCK_SIZE - 1) / BLOCK_SIZE; ++m) {

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        GetSubMatrix(A, &As[0][0], numACols, blockRow, m, row, col);
        GetSubMatrix(B, &Bs[0][0], numBCols, m, blockCol, row, col);

              // Synchronize to make sure the sub-matrices are loaded
              // before starting the computation
            __syncthreads();

            // Multiply Asub and Bsub together
            for (int e = 0; e < BLOCK_SIZE; ++e) {
                  Cvalue += As[row][e] * Bs[e][col];
                }

            // Synchronize to make sure that the preceding
            // computation is done before loading two new
            // sub-matrices of A and B in the next iteration
            __syncthreads();
        }

      // Write Csub to device memory
      // Each thread writes one element
          if ((blockRow * BLOCK_SIZE + row) < numARows && (blockCol * BLOCK_SIZE + col) < numBCols) {
              SetElement(C, numBCols, blockRow * BLOCK_SIZE + row, blockCol * BLOCK_SIZE + col, Cvalue);
              // C[(blockRow * BLOCK_SIZE + row) * numBCols + (blockCol * BLOCK_SIZE + col)] = Cvalue;
            }
    #endif

}

void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        cout << msg << ": " << hipGetErrorString(err) << endl;
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char *argv[])
{
    // Controllo dei parametri da riga di comando
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <threadSizeX> <threadSizeY>" << std::endl;
        return 1;
    }

    int threadSizeX = std::atoi(argv[1]);
    int threadSizeY = std::atoi(argv[2]);

    // Dimensioni della matrice
    const int r1 = ROW1, c1 = COL1, r2 = ROW2, c2 = COL2;

    #ifdef USE_SHARED_MEMORY
        size_t pitchA = 0, pitchB = 0, pitchC = 0;

        // Vector A,B, Mult
        vector<int> a(r1 * c1); // Matrice A
        vector<int> b(r2 * c2); // Matrice B
        vector<int> mult(r1 * c2, 0); // Matrice risultato

            // Init randomly the matrixes
        for (int i = 0; i < r1; ++i)
            for (int j = 0; j < c1; ++j)
            {
                a[i * c1 + j] = rand() % 10;
            }

        for (int i = 0; i < r2; ++i)
            for (int j = 0; j < c2; ++j)
            {
                b[i * c2 + j] = rand() % 10;
            }

        for (int i = 0; i < r1; ++i)
            for (int j = 0; j < c2; ++j)
            {
                mult[i * c2 + j] = 0;
            }

        // cout << "this is matrix A: " << endl;

        //print matrix A
        // for (int i = 0; i < r1; ++i)
        // {
        //     for (int j = 0; j < c1; ++j)
        //     {
        //         cout << a[i * c1 + j] << " ";
        //     }
        //     cout << endl;
        // }

        // cout << "this is matrix B: " << endl;

        //print matrix B
        // for (int i = 0; i < r1; ++i)
        // {
        //     for (int j = 0; j < c1; ++j)
        //     {
        //         cout << b[i * c1 + j] << " ";
        //     }
        //     cout << endl;
        // }

    #endif



    // If column of first matrix in not equal to row of second matrix,
    while (c1!=r2)
        {
            cout << "Error! Column of first matrix not equal to row of second.";
        }


//------------------------------------------------------------------------------------------------------------------------------------------------------------------------------

// Parameter for the kernel function

// Allocate device memory
    int *d_mult, *d_a, *d_b;

// Configuration

    #ifdef USE_SHARED_MEMORY
    // Creating space in the GPU (checkCudaError for handling the error)
        checkCudaError(hipMalloc(&d_a, r1 * c1 * sizeof(int)), "Errore allocazione memoria per d_a");
        checkCudaError(hipGetLastError(), "Error after hipMemcpy d_a");

        checkCudaError(hipMalloc(&d_b, r2 * c2 * sizeof(int)), "Errore allocazione memoria per d_b");
        checkCudaError(hipGetLastError(), "Error after hipMemcpy d_b");

        checkCudaError(hipMalloc(&d_mult, r1 * c2 * sizeof(int)), "Errore allocazione memoria per d_mult");
        checkCudaError(hipGetLastError(), "Error after hipMemcpy d_mult");

        checkCudaError(hipMemcpy(d_a, a.data(), r1 * c1 * sizeof(int), hipMemcpyHostToDevice), "Errore copia dati per d_a");
        checkCudaError(hipMemcpy(d_b, b.data(), r2 * c2 * sizeof(int), hipMemcpyHostToDevice), "Errore copia dati per d_b");
        checkCudaError(hipMemcpy(d_mult, mult.data(), r1 * c2 * sizeof(int), hipMemcpyHostToDevice), "Errore copia dati per d_mult");
        hipDeviceSynchronize();
    #endif

//----------------------Check the correctness--------------

// Check the correctness of copy

    // Verifica che i dati siano copiati correttamente dalla CPU alla GPU e viceversa

        int *h_a_check = new int[r1 * c1];
        int *h_b_check = new int[r2 * c2];
        int *h_mult_check = new int[r1 * c2];
        bool is_correct =  true;

    #ifdef USE_SHARED_MEMORY

        checkCudaError(hipMemcpy(h_a_check, d_a, r1 * c1 * sizeof(int), hipMemcpyDeviceToHost), "Errore copia dati da d_a");
        checkCudaError(hipMemcpy(h_b_check, d_b, r2 * c2 * sizeof(int), hipMemcpyDeviceToHost), "Errore copia dati da d_b");
        checkCudaError(hipMemcpy(h_mult_check, d_mult, r1 * c2 * sizeof(int), hipMemcpyDeviceToHost), "Errore copia dati da d_mult");
        hipDeviceSynchronize();

        // Verifica per la matrice A
        for (int i = 0; i < r1 * c1; ++i) {
            if (a[i] != h_a_check[i]) {
                cout << "Errore nella copia dei dati per la matrice A all'elemento " << i
                    << ": atteso " << a[i] << ", ottenuto " << h_a_check[i] << endl;
                is_correct = false;
            }
        }

        // Verifica per la matrice B
        for (int i = 0; i < r2 * c2; ++i) {
            if (b[i] != h_b_check[i]) {
                cout << "Errore nella copia dei dati per la matrice B all'elemento " << i
                    << ": atteso " << b[i] << ", ottenuto " << h_b_check[i] << endl;
                is_correct = false;
            }
        }

        if (is_correct) {
            cout << "Dati copiati correttamente." << endl;
        } else {
            cout << "Errore nella copia dei dati." << endl;
        }

    #endif


// --------------------- Configuration of # Threads and # Block for each implementation --------------------------------------------------------------------------

    #ifdef USE_SHARED_MEMORY
        dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
        dim3 gridSize((c2 + blockSize.x - 1) / blockSize.x, (r1 + blockSize.y - 1) / blockSize.y);

    #endif

    /*
    int totalThreads = blockSize.x * blockSize.y * blockSize.z * threadSize.x * threadSize.y * threadSize.z;
    int totalElements = r1 * c2;
    cout << "threadSize: (" << threadSize.x << ", " << threadSize.y << ", " << threadSize.z << ")" <<endl;
    */
    cout << "blockSize: (" << blockSize.x << ", " << blockSize.y << ", " << blockSize.z << ")" <<endl;
    // cout << "Total threads: " << totalThreads <<endl;
    // cout << "Total threads for each block: " << threadSize.x * threadSize.y * threadSize.z <<endl;
    cout << "Total number of blocks (must be less or equal than 1024): " << blockSize.x * blockSize.y * blockSize.z  <<endl;

    // Timing
    // Usa hipEvent_t per misurare il tempo CUDA
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "Failed to create start event");
    checkCudaError(hipEventCreate(&stop), "Failed to create stop event");

    // Avvia il timer
    checkCudaError(hipEventRecord(start, 0), "Failed to record start event");

    // Lunch kernel function
    // matrixMultiply<<<threadSize,blockSize>>>(d_a, d_b, d_mult, r1, c1, c2, pitchA, pitchB, pitchC);
        matrixMultiply<<<gridSize, blockSize>>>(d_a, d_b, d_mult, r1, c1, c2, pitchA, pitchB, pitchC);
        checkCudaError(hipGetLastError(), "Error after kernel launch");
        hipDeviceSynchronize();

    // Ferma il timer
    checkCudaError(hipEventRecord(stop, 0), "Failed to record stop event");
    checkCudaError(hipEventSynchronize(stop), "Failed to synchronize stop event");

    float milliseconds = 0;
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop), "Failed to calculate elapsed time");

    std::cout << "Time elapsed: " << milliseconds << " milliseconds." << std::endl;

    #ifdef USE_SHARED_MEMORY
        // Copy result from device to host
        hipMemcpy(mult.data(), d_mult, r1 * c2 * sizeof(int), hipMemcpyDeviceToHost);
        checkCudaError(hipGetLastError(), "Error after hipMemcpy d_mult");
    #endif


    // ------------------------------------------------------------

    // Creazione di matrici Eigen utilizzando i dati esistenti
    // Eigen::MatrixXi matrix1(r1, c1);
    // Eigen::MatrixXi matrix2(r2, c2);


    // for (int i = 0; i < r1; ++i) {
    //     for (int j = 0; j < c1; ++j) {

    //         #ifdef USE_SHARED_MEMORY
    //             matrix1(i, j) = a[i * c1 + j];
    //         #endif
    //     }
    // }

    // for (int i = 0; i < r2; ++i) {
    //     for (int j = 0; j < c2; ++j) {

    //             #ifdef USE_SHARED_MEMORY
    //                 matrix2(i, j) = b[i * c2 + j];
    //             #endif
    //     }
    // }

    // Calcolo del prodotto tra le due matrici
    // Eigen::MatrixXd result1 = matrix1.cast<double>() * matrix2.cast<double>();

    // Creazione della matrice mult_double per il confronto
    // Eigen::MatrixXd mult_double(r1, c2);
    // for (int i = 0; i < r1; ++i) {
    //     for (int j = 0; j < c2; ++j) {

    //         #ifdef USE_SHARED_MEMORY
    //             mult_double(i, j) = mult[i * c2 + j];
    //         #endif

    //     }
    // }


    // Stampa dei risultati
    //cout << "Result with Eigen:" << endl << result1 << endl;
    //cout << "Result with program:" << endl << mult_double << endl;

    // Confronto tra le due matrici
    // if (result1.isApprox(mult_double, 1e-5)) {
    //     cout << "Matrices are equal." << endl;
    // } else {
    //     cout << "Matrices are not equal." << endl;
    // }

    // Free device memory
    hipFree(d_mult);
    hipFree(d_a);
    hipFree(d_b);
    delete[] h_a_check;
    delete[] h_b_check;
    delete[] h_mult_check;

    return 0;
}